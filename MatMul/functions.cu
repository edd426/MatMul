#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <algorithm>

using namespace std;


__global__ void cudamult(int n, int p, int m, float* A, float* B, float* C)
{
	//int index = threadIdx.x;

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	float tmpSum = 0;

	if (ROW > n || COL > m)
		return;
	// each thread computes one element of the block sub-matrix
	for (int i = 0; i < p; ++i) {
		tmpSum += A[ROW * p + i] * B[i * m + COL];
	}

	printf("ROW:%d COL:%d sum: %d \n", ROW, COL, tmpSum); // DEBUG

	C[ROW * m + COL] = tmpSum;

}

extern void use_cudamult(int n, int p, int m, float* A, float* B, float* C)
{
	//int threadsPerBlock = 512;
	//int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
	int numThreads = max(max(n, p), m);
	dim3 threadsPerBlock(numThreads, numThreads);
	dim3 blocksPerGrid(1, 1);
	//threadsPerBlock.x = 512;
	//threadsPerBlock.y = 512;
	//blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
	//blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));


	cudamult << <blocksPerGrid, threadsPerBlock >> > (n, p, m, A, B, C);
	hipDeviceSynchronize();
}